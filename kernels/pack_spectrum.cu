#include "hip/hip_runtime.h"
#include "../include/pack_spectrum.h"

__device__ float2 permute(float2 data, int2 id) {
    float sign = (1.0f - 2.0f * ((id.x + id.y) % 2));
    return make_float2(data.x * sign, data.y * sign);
}

__global__ void assemble_maps(float2* dx_dz, float2* dy_dxz, float2* dyx_dyz, float2* dxx_dzz,
    float3* displacement, float2* slope, int N, float2 lambda) {
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N && j < N) {
        int idx = i * N + j;
        float2 htilde_displacement_X = permute(dx_dz[idx], make_int2(j, i));
        float2 htilde_displacement_Z = permute(dy_dxz[idx], make_int2(j, i));
        float2 htilde_slope_X = permute(dyx_dyz[idx], make_int2(j, i));
        float2 htilde_slope_Z = permute(dxx_dzz[idx], make_int2(j, i));

        float slopeX = htilde_slope_X.x / (1 + fabsf(htilde_slope_Z.x * lambda.x));
        float slopeY = htilde_slope_X.y / (1 + fabsf(htilde_slope_Z.y * lambda.y));

        displacement[idx] = make_float3(lambda.x * htilde_displacement_X.x,
            htilde_displacement_Z.x, lambda.y * htilde_displacement_X.y);
        slope[idx] = make_float2(slopeX, slopeY);       
    }
}

__global__ void apply_brightness_kernel(float2* h0_k, float brightness, int N) {
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N && j < N) {
        int idx = i * N + j;
        h0_k[idx].x *= brightness;
        h0_k[idx].y *= brightness;

        h0_k[idx].x = fmaxf(0.0f, fminf(255.0f, h0_k[idx].x * 255.0f));
        h0_k[idx].y = fmaxf(0.0f, fminf(255.0f, h0_k[idx].y * 255.0f));
    }
}

void pack_and_assemble(float2* dx_dz, float2* dy_dxz, float2* dyx_dyz, float2* dxx_dzz,
    float3* displacement, float2* slope, int N, float2 lambda) {
    dim3 blockDim(32, 32);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    assemble_maps<<<gridDim, blockDim>>>(dx_dz, dy_dxz, dyx_dyz, dxx_dzz, displacement, slope, N, lambda);
    hipDeviceSynchronize();
}

void apply_brightness(float2* h0_k, float brightness, int N) {
    dim3 blockDim(32, 32);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    apply_brightness_kernel<<<gridDim, blockDim>>>(h0_k, brightness, N);
    hipDeviceSynchronize();
}